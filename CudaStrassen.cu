
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <bits/stdc++.h>

using namespace std;

void print(int n, int** mat)
{
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            cout << mat[i][j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

int* assignMatrix(int n)
{
    int* data = (int*)malloc(n * n * sizeof(int));
    return data;
}

int** assignMatrix2D(int n)
{
    int* data = (int*)malloc(n * n * sizeof(int));
    int** array = (int**)malloc(n * sizeof(int*));
    for (int i = 0; i < n; i++)
    {
        array[i] = &(data[n * i]);
    }
    return array;
}

void populateMat(int n, int*& mat)
{
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            mat[i * n + j] = rand() % 5;
        }
    }
}

void populateMat2D(int n, int** &mat)
{
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            mat[i][j] = rand() % 5;
        }
    }
}

int** openSL(int n, int** mat, int offseti, int offsetj)
{
    int m = n / 2;
    int** slice = assignMatrix2D(m);
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < m; j++)
        {
            slice[i][j] = mat[offseti + i][offsetj + j];
        }
    }
    return slice;
}

int** MatSum(int n, int** mat1, int** mat2, bool add)
{
    int** result = assignMatrix2D(n);
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (add)
                result[i][j] = mat1[i][j] + mat2[i][j];
            else
                result[i][j] = mat1[i][j] - mat2[i][j];
        }
    }

    return result;
}

int** MatJoin(int m, int** c11, int** c12, int** c21, int** c22)
{
    int n = 2 * m;
    int** result = assignMatrix2D(n);

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i < m && j < m)
                result[i][j] = c11[i][j];
            else if (i < m)
                result[i][j] = c12[i][j - m];
            else if (j < m)
                result[i][j] = c21[i - m][j];
            else
                result[i][j] = c22[i - m][j - m];
        }
    }

    return result;
}

void MatrixClear(int n, int* mat)
{
    free(mat);
}

void MatrixClear2D(int n, int** mat)
{
    free(mat[0]);
    free(mat);
}

__global__ void multiply(int* mat1, int* mat2, int* product, int n)
{
    int prod = blockIdx.x * blockDim.x + threadIdx.x;
    int i = prod / n;
    int j = prod % n;
    for (int k = 0; k < n; k++) {
        product[i * n + j] += mat1[i * n + k] * mat2[k * n + j];
    }
}

int** cudaNaive(int n, int** mat1, int** mat2)
{
    int* h_mat1 = assignMatrix(n);
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            h_mat1[i*n + j] = mat1[i][j];
        }
    }

    int* h_mat2 = assignMatrix(n);
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            h_mat2[i*n + j] = mat2[i][j];
        }
    }

    int* h_product = assignMatrix(n);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            h_product[i * n + j] = 0;
        }
    }

    size_t bytes = n * n * sizeof(int);

    int *d_mat1, *d_mat2, *d_product;

    hipMalloc(&d_mat1, bytes);
    hipMalloc(&d_mat2, bytes);
    hipMalloc(&d_product, bytes);

    hipMemcpy(d_mat1, h_mat1, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, h_mat2, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_product, h_product, bytes, hipMemcpyHostToDevice);

    int threads = min(1024, n);
    int blocks = (n * n) / threads;
    dim3 gridSize(blocks, 1, 1);
    dim3 blockSize(threads, 1, 1);

    multiply<<<gridSize, blockSize>>>(d_mat1, d_mat2, d_product, n);
    hipDeviceSynchronize();

    hipMemcpy(h_product, d_product, bytes, hipMemcpyDeviceToHost);

    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_product);

    MatrixClear(n, h_mat1);
    MatrixClear(n, h_mat2);

    int** product = assignMatrix2D(n);
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            product[i][j] = h_product[i*n + j];
        }
    }
    return product;
}


int** strassen(int n, int** mat1, int** mat2)
{

    int m = n / 2;

    int** a = openSL(n, mat1, 0, 0);
    int** b = openSL(n, mat1, 0, m);
    int** c = openSL(n, mat1, m, 0);
    int** d = openSL(n, mat1, m, m);
    int** e = openSL(n, mat2, 0, 0);
    int** f = openSL(n, mat2, 0, m);
    int** g = openSL(n, mat2, m, 0);
    int** h = openSL(n, mat2, m, m);

    int** bds = MatSum(m, b, d, false);
    int** gha = MatSum(m, g, h, true);
    int** s1 = cudaNaive(m, bds, gha);
    MatrixClear2D(m, bds);
    MatrixClear2D(m, gha);

    int** ada = MatSum(m, a, d, true);
    int** eha = MatSum(m, e, h, true);
    int** s2 = cudaNaive(m, ada, eha);
    MatrixClear2D(m, ada);
    MatrixClear2D(m, eha);

    int** acs = MatSum(m, a, c, false);
    int** efa = MatSum(m, e, f, true);
    int** s3 = cudaNaive(m, acs, efa);
    MatrixClear2D(m, acs);
    MatrixClear2D(m, efa);

    int** aba = MatSum(m, a, b, true);
    int** s4 = cudaNaive(m, aba, h);
    MatrixClear2D(m, aba);
    MatrixClear2D(m, b);

    int** fhs = MatSum(m, f, h, false);
    int** s5 = cudaNaive(m, a, fhs);
    MatrixClear2D(m, fhs);
    MatrixClear2D(m, a);
    MatrixClear2D(m, f);
    MatrixClear2D(m, h);

    int** ges = MatSum(m, g, e, false);
    int** s6 = cudaNaive(m, d, ges);
    MatrixClear2D(m, ges);
    MatrixClear2D(m, g);

    int** cda = MatSum(m, c, d, true);
    int** s7 = cudaNaive(m, cda, e);
    MatrixClear2D(m, cda);
    MatrixClear2D(m, c);
    MatrixClear2D(m, d);
    MatrixClear2D(m, e);

    int** s1s2a = MatSum(m, s1, s2, true);
    int** s6s4s = MatSum(m, s6, s4, false);
    int** c11 = MatSum(m, s1s2a, s6s4s, true);
    MatrixClear2D(m, s1s2a);
    MatrixClear2D(m, s6s4s);
    MatrixClear2D(m, s1);

    int** c12 = MatSum(m, s4, s5, true);
    MatrixClear2D(m, s4);

    int** c21 = MatSum(m, s6, s7, true);
    MatrixClear2D(m, s6);

    int** s2s3s = MatSum(m, s2, s3, false);
    int** s5s7s = MatSum(m, s5, s7, false);
    int** c22 = MatSum(m, s2s3s, s5s7s, true);
    MatrixClear2D(m, s2s3s);
    MatrixClear2D(m, s5s7s);
    MatrixClear2D(m, s2);
    MatrixClear2D(m, s3);
    MatrixClear2D(m, s5);
    MatrixClear2D(m, s7);

    int** prod = MatJoin(m, c11, c12, c21, c22);

    MatrixClear2D(m, c11);
    MatrixClear2D(m, c12);
    MatrixClear2D(m, c21);
    MatrixClear2D(m, c22);

    return prod;
}

int main()
{
    int n;
    cout << "\nEnter Matrix Size in power of 2 (2,4,8,16,32,64,128,256,....16384): ";
    cin >> n;

    //n = 1024

    int** mat1 = assignMatrix2D(n);
    populateMat2D(n, mat1);

    int** mat2 = assignMatrix2D(n);
    populateMat2D(n, mat2);

    clock_t start, end;
    start = clock();

    int** prod = strassen(n, mat1, mat2);

    end = clock();
    double time = double(end - start) / double(CLOCKS_PER_SEC);
    cout << endl;
    cout << "\nParallel CUDA runtime with matrix " << n << " X " << n << " is: " << time;
    cout << endl;
    cout << endl;
    return 0;
}
